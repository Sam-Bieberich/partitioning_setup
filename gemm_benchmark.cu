#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <iostream>
#include <vector>
#include <chrono>

// From Cody

// Set default precision type
using Real = float; // Change to double for double precision
#define GEMM hipblasSgemm // Change to cublasDgemm for double precision

double elapsed_sec(hipEvent_t start, hipEvent_t stop) {
    float ms = 0;
    hipEventElapsedTime(&ms, start, stop);
    return ms / 1000.0;
}

int main(int argc, char** argv) {
    // Allow matrix size configuration
    int size = 4096; // Default: 4096x4096
    int iters = 10;  // Default: 10 iterations
    
    if (argc > 1) size = atoi(argv[1]);
    if (argc > 2) iters = atoi(argv[2]);
    
    int m = size, n = size, k = size;
    Real alpha = 1.0, beta = 0.0;
    
    std::cout << "=== GEMM Benchmark ===" << std::endl;
    std::cout << "Matrix size: " << m << "x" << n << "x" << k << std::endl;
    std::cout << "Iterations: " << iters << std::endl;
    std::cout << "Precision: " << (sizeof(Real) == 4 ? "Single (FP32)" : "Double (FP64)") << std::endl;
    
    // Check CUDA device
    int device;
    hipGetDevice(&device);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device);
    std::cout << "GPU: " << prop.name << std::endl;
    std::cout << "Memory: " << prop.totalGlobalMem / (1024*1024) << " MB" << std::endl;
    std::cout << std::endl;
    
    // Allocate host memory
    std::vector<Real> h_A(m * k, 1.0);
    std::vector<Real> h_B(k * n, 2.0);
    std::vector<Real> h_C(m * n, 0.0);
    
    // Allocate device memory
    Real *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, m * k * sizeof(Real));
    hipMalloc((void**)&d_B, k * n * sizeof(Real));
    hipMalloc((void**)&d_C, m * n * sizeof(Real));
    
    // Copy data to device
    hipMemcpy(d_A, h_A.data(), m * k * sizeof(Real), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B.data(), k * n * sizeof(Real), hipMemcpyHostToDevice);
    
    // Create cuBLAS handle
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    
    // Warmup run
    std::cout << "Warming up..." << std::endl;
    GEMM(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
         m, n, k, &alpha, d_A, m, d_B, k, &beta, d_C, m);
    hipDeviceSynchronize();
    
    // Timed runs
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    std::cout << "Running benchmark..." << std::endl;
    hipEventRecord(start);
    
    for (int i = 0; i < iters; i++) {
        // GEMM: C = alpha*A*B + beta*C
        // A: (m x k), B: (k x n), C: (m x n)
        GEMM(
            handle,
            HIPBLAS_OP_N, HIPBLAS_OP_N, // no transpose for A and B
            m, n, k,
            &alpha,
            d_A, m,
            d_B, k,
            &beta,
            d_C, m
        );
    }
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    double seconds = elapsed_sec(start, stop);
    
    // Copy result back
    hipMemcpy(h_C.data(), d_C, m * n * sizeof(Real), hipMemcpyDeviceToHost);
    
    // Compute and print FLOPS
    double flops = iters * 2.0 * m * n * k; // GEMM FLOPS formula
    double gflops = flops / seconds / 1e9;
    double avg_time_ms = (seconds / iters) * 1000.0;
    
    std::cout << std::endl;
    std::cout << "=== Results ===" << std::endl;
    std::cout << "Total time: " << seconds << " seconds" << std::endl;
    std::cout << "Avg time per iteration: " << avg_time_ms << " ms" << std::endl;
    std::cout << "Performance: " << gflops << " GFLOPS" << std::endl;
    std::cout << "Verification C[0] = " << h_C[0] << " (expected: " << (2.0 * k) << ")" << std::endl;
    
    // Cleanup
    hipblasDestroy(handle);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    
    return 0;
}